#include "hip/hip_runtime.h"
#include "labeling.h"
#include <cstdio>
#include <cassert>

#define MAXSIZE (40 << 20)
#define THREAD_OF_BLOCK 1024

namespace lab2 {
    __device__ int segscan_warp(int *ptr, bool *flag, const unsigned int tid) {
        const unsigned int lane = tid & 31;
        if (lane >= 1) {
            ptr[tid] = flag[tid] ? ptr[tid] : ptr[tid-1] + ptr[tid];
            flag[tid] = flag[tid-1] | flag[tid];
        }
        if (lane >= 2) {
            ptr[tid] = flag[tid] ? ptr[tid] : ptr[tid-2] + ptr[tid];
            flag[tid] = flag[tid-2] | flag[tid];
        }
        if (lane >= 4) {
            ptr[tid] = flag[tid] ? ptr[tid] : ptr[tid-4] + ptr[tid];
            flag[tid] = flag[tid-4] | flag[tid];
        }
        if (lane >= 8) {
            ptr[tid] = flag[tid] ? ptr[tid] : ptr[tid-8] + ptr[tid];
            flag[tid] = flag[tid-8] | flag[tid];
        }
        if (lane >= 16) {
            ptr[tid] = flag[tid] ? ptr[tid] : ptr[tid-16] + ptr[tid];
            flag[tid] = flag[tid-16] | flag[tid];
        }
        return ptr[tid];
    }

    __device__ int segscan_block(int *ptr, bool *flag, const unsigned int tid) {
        unsigned int warpid = tid >> 5;
        unsigned int warp_first = warpid << 5;
        unsigned int warp_last = warp_first + 31;

        bool warp_is_open = (flag[warp_first] == 0);
        __syncthreads();

        int val = segscan_warp(ptr, flag, tid);
        int warp_total = ptr[warp_last];
        int warp_flag = (flag[warp_last] != 0 || !warp_is_open);
        bool will_accumulate = (warp_is_open && flag[tid] == 0);
        __syncthreads();

        if (tid == warp_last) {
            ptr[warpid] = warp_total;
            flag[warpid] = warp_flag;
        }
        __syncthreads();

        if (warpid == 0) {
            segscan_warp(ptr, flag, tid);
        }
        __syncthreads();

        if (warpid != 0 && will_accumulate) {
            val = ptr[warpid-1] + val;
        }
        __syncthreads();

        ptr[tid] = val;
        __syncthreads();

        return val;
    }

    __global__ void segscan(const char *i_data, int *o_data, int data_size) {
        __shared__ int val[THREAD_OF_BLOCK];
        __shared__ bool flag[THREAD_OF_BLOCK];
        const unsigned int tid = threadIdx.x;
        const unsigned int base = blockIdx.x * blockDim.x * 4 + 4 * tid;

        // Handle 4 elements in global device memory
        int sub_val[4] = { 0 };
        for (int i = 0; i < 4 && (base + i) < data_size; i++) {
            sub_val[i] = (i_data[base + i] == ' ') ? 0 : 1;
        }
        bool sub_flag[4] = {1, 1, 1, 1};
        for (int i = 0; i < 4 && (base + i) < data_size; i++) {
            if (i_data[base + i] != ' ' && (base + i > 0) && i_data[base + i - 1] != ' ') {
                sub_flag[i] = 0;
            }
        }
        __syncthreads();

        // Segscan 4 elements sequentially
        for (int i = 1; i < 4; i++) {
            sub_val[i] = sub_flag[i] ? sub_val[i] : sub_val[i-1] + sub_val[i];
            sub_flag[i] = sub_flag[i-1] | sub_flag[i];
        }
        val[tid] = sub_val[3];
        flag[tid] = sub_flag[3];
        __syncthreads();

        // Intra-block scan
        int ret = segscan_block(val, flag, tid);
        __syncthreads();

        // Accumulate
        if (tid != 0) {
            for (int i = 0; i < 4; i++) {
                if (sub_flag[0] == 0 && sub_flag[i] == 0) {
                    sub_val[i] += val[tid - 1];
                }
            }
        }
        __syncthreads();

        // Write back
        for (int i = 0; i < 4 && (base + i) < data_size; i++) {
            o_data[base + i] = sub_val[i];
        }
    }

    __global__ void sweep(int *data, int data_size) {
        const unsigned int base = (blockIdx.x + 1) * (THREAD_OF_BLOCK * 4);
        const unsigned int tid = threadIdx.x;
        if (base + tid < data_size) {
            if (data[base + tid] - 1 == tid) {
                data[base + tid] += data[base - 1];
            }
        }
    }
}

void labeling(const char *cuStr, int *cuPos, int strLen){
    int nb_blocks = ((strLen-1)/THREAD_OF_BLOCK) / 4 + 1;
    lab2::segscan<<<nb_blocks, THREAD_OF_BLOCK>>>(cuStr, cuPos, strLen);
    lab2::sweep<<<nb_blocks-1, THREAD_OF_BLOCK>>>(cuPos, strLen);
}
